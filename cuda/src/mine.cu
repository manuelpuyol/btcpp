#include "hip/hip_runtime.h"
#include<mine.cuh>

__global__ void mine(BYTE *in, BYTE *out, int *result, uint32_t *nonce, int size, int difficulty) {
  int id = (blockIdx.x * blockDim.x) + threadIdx.x;

  uint32_t test = id * BUCKET;
  uint32_t end = (id + 1) * BUCKET;

  if(id == TOTAL - 1) {
    end = UINT32_MAX;
  }

  csha256(in, out, size);
  if(verify(out, difficulty, result)) {
    *nonce = test;
  }
}

uint32_t cmine(string str, int difficulty) {
  // host
  BYTE *buff = reinterpret_cast<unsigned char*>(const_cast<char*>(str.c_str()));
  int size = str.length();
  int res = NOT_FOUND;
  uint32_t n;

  // device
  BYTE *in, *out;
  int *result;
  uint32_t *nonce;

  hipMalloc((void **)&in, size);
  hipMalloc((void **)&out, SHA256_BLOCK_SIZE);
  hipMalloc((void **)&result, sizeof(int));
  hipMalloc((void **)&nonce, sizeof(uint32_t));

  hipMemcpy(in, buff, size * sizeof(BYTE), hipMemcpyHostToDevice);
  hipMemcpy(result, &res, sizeof(int), hipMemcpyHostToDevice);

  pre_sha256();
  mine<<< 1, 1 >>>(in, out, result, nonce, size, difficulty);

  hipDeviceSynchronize();

  hipMemcpy(&res, result, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&n, nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);

  cout << "result = " << res << endl;
  cout << "nonce = " << n << endl;

  hipFree(in);
  hipFree(out);
  hipFree(result);
  hipFree(nonce);

  return n;
}
