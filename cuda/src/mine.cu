#include "hip/hip_runtime.h"
#include<mine.cuh>

__global__ void mine(BYTE *in, BYTE *out, int *result, int size, int difficulty) {
  int id = (blockIdx.x * blockDim.x) + threadIdx.x;

  uint32_t test = id * BUCKET;
  uint32_t end = (id + 1) * BUCKET;

  if(id == TOTAL - 1) {
    end = UINT32_MAX;
  }

  printf("id = %d - start = %" PRIu32 " - end = %" PRIu32 "\n ", id, test, end);
  // csha256(in, out, size);
  // verify(out, difficulty, result);
}

void cmine(string str, int difficulty) {
  // host
  BYTE *buff = reinterpret_cast<unsigned char*>(const_cast<char*>(str.c_str()));
  int size = str.length();
  int res = NOT_FOUND;

  // device
  BYTE *in, *out;
  int *result;

  hipMalloc((void **)&in, size);
  hipMalloc((void **)&out, SHA256_BLOCK_SIZE);
  hipMalloc((void **)&result, sizeof(int));

  hipMemcpy(in, buff, size * sizeof(BYTE), hipMemcpyHostToDevice);
  hipMemcpy(result, &res, sizeof(int), hipMemcpyHostToDevice);

  pre_sha256();
  mine <<< BLOCKS, THREADS >>> (in, out, result, size, difficulty );

  hipDeviceSynchronize();

  hipMemcpy(&res, result, sizeof(int), hipMemcpyDeviceToHost);

  cout << "result = " << res << endl;

  hipFree(in);
  hipFree(out);
  hipFree(result);
}