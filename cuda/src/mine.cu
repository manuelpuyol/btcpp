#include "hip/hip_runtime.h"
#include<mine.cuh>

__global__ void mine(BYTE *in, int *result, uint32_t *nonce, int size, int difficulty) {
  int id = (blockIdx.x * blockDim.x) + threadIdx.x;

  uint32_t ntest = id * BUCKET;
  uint32_t end = (id + 1) * BUCKET;

  if(id == TOTAL - 1) {
    end = UINT32_MAX;
  }

  BYTE *btest = (BYTE *) malloc(size);
  BYTE *hash = (BYTE *) malloc(SHA256_BLOCK_SIZE);

  memcpy(btest, in, size);

  csha256(btest, hash, size);
  if(verify(hash, difficulty, result)) {
    *nonce = ntest;
  }
}

uint32_t cmine(string str, int difficulty) {
  // host
  BYTE *buff = reinterpret_cast<unsigned char*>(const_cast<char*>(str.c_str()));
  int size = str.length();
  int res = NOT_FOUND;
  uint32_t n;

  // device
  BYTE *in;
  int *result;
  uint32_t *nonce;

  hipMalloc((void **)&in, size);
  hipMalloc((void **)&result, sizeof(int));
  hipMalloc((void **)&nonce, sizeof(uint32_t));

  hipMemcpy(in, buff, size * sizeof(BYTE), hipMemcpyHostToDevice);
  hipMemcpy(result, &res, sizeof(int), hipMemcpyHostToDevice);

  pre_sha256();
  mine<<< 1, 1 >>>(in, result, nonce, size, difficulty);

  hipDeviceSynchronize();

  hipMemcpy(&res, result, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&n, nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);

  cout << "result = " << res << endl;
  cout << "nonce = " << n << endl;

  hipFree(in);
  hipFree(result);
  hipFree(nonce);

  return n;
}
