#include "hip/hip_runtime.h"
#include<mine.cuh>

__global__ void mine(BYTE *in, int *result, uint32_t *nonce, int size, int difficulty) {
  int id = (blockIdx.x * blockDim.x) + threadIdx.x;

  uint32_t test = 12345;
  uint32_t end = (id + 1) * BUCKET;

  if(id == TOTAL - 1) {
    end = UINT32_MAX;
  }

  if(verify(in, test, size, difficulty, result)) {
    *nonce = test;
  }
}

uint32_t cmine(string str, int difficulty) {
  // host
  BYTE *buff = reinterpret_cast<unsigned char*>(const_cast<char*>(str.c_str()));
  int size = str.length();
  int res = NOT_FOUND;
  uint32_t n;

  // device
  BYTE *in;
  int *result;
  uint32_t *nonce;
  
  uint32_t x = 1234;
  int length = snprintf( NULL, 0, "%" PRIu32, x );

  hipMalloc((void **)&in, size);
  hipMalloc((void **)&result, sizeof(int));
  hipMalloc((void **)&nonce, sizeof(uint32_t));

  hipMemcpy(in, buff, size * sizeof(BYTE), hipMemcpyHostToDevice);
  hipMemcpy(result, &res, sizeof(int), hipMemcpyHostToDevice);

  pre_sha256();
  mine<<< 1, 1 >>>(in, result, nonce, size, difficulty);

  hipDeviceSynchronize();

  hipMemcpy(&res, result, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&n, nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);

  cout << "result = " << res << endl;
  cout << "nonce = " << n << endl;

  hipFree(in);
  hipFree(result);
  hipFree(nonce);

  return n;
}
