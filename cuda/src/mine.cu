#include "hip/hip_runtime.h"
#include<mine.cuh>

__global__ void mine(BYTE *in, BYTE *out, int size, int difficulty, int *result) {
  int id = (blockIdx.x * blockDim.x) + threadIdx.x;

  printf("id = %d\n ", id);
  // csha256(in, out, size);
  // verify(out, difficulty, result);
}

void cmine(string str, int difficulty) {
  BYTE *buff = reinterpret_cast<unsigned char*>(const_cast<char*>(str.c_str()));

  int size = str.length();
  int res = NOT_FOUND;
  BYTE *in, *out;
  int *result;

  hipMalloc((void **)&in, size);
  hipMalloc((void **)&out, SHA256_BLOCK_SIZE);
  hipMalloc((void **)&result, sizeof(int));

  hipMemcpy(in, buff, size * sizeof(BYTE), hipMemcpyHostToDevice);
  hipMemcpy(result, &res, sizeof(int), hipMemcpyHostToDevice);

  pre_sha256();
  mine <<< BLOCKS, THREADS >>> (in, out, size, difficulty, result);

  hipDeviceSynchronize();

  hipMemcpy(&res, result, sizeof(int), hipMemcpyDeviceToHost);

  cout << "result = " << res << endl;

  hipFree(in);
  hipFree(out);
  hipFree(result);
}