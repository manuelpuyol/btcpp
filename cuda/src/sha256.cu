#include "hip/hip_runtime.h"
// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list

#include<sha256.cuh>

__global__ void sha256_cuda(JOB *job) {
  // perform sha256 calculation here
  SHA256_CTX ctx;
  sha256_init(&ctx);
  sha256_update(&ctx, job->data, job->size);
  sha256_final(&ctx, job->digest);
}

void pre_sha256() {
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}


void runJob(JOB *job){
  sha256_cuda <<< 1, 1 >>> (job);
}


JOB *JOB_init(BYTE *data, long size) {
  JOB *j;
  checkCudaErrors(hipMallocManaged(&j, sizeof(JOB)));	//j = (JOB *)malloc(sizeof(JOB));
  checkCudaErrors(hipMallocManaged(&(j->data), size));
  j->data = data;
  j->size = size;
  for (int i = 0; i < 64; i++) {
    j->digest[i] = 0xff;
  }

  return j;
}


void cuda_sha256() {
  BYTE *buff = (BYTE *) "lol";
  JOB *job;

  checkCudaErrors(hipMallocManaged(&job, sizeof(JOB)));
  job = JOB_init(buff, sizeof(buff));

  pre_sha256();
  runJob(job);

  print_job(job);

  hipDeviceSynchronize();
  hipDeviceReset();
}