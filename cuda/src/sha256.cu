#include "hip/hip_runtime.h"
#include<sha256.cuh>

__global__ void sha256_cuda(BYTE *in, BYTE *out, int size, int difficulty, int *result) {
  SHA256_CTX ctx;
  sha256_init(&ctx);
  sha256_update(&ctx, in, size);
  sha256_final(&ctx, out);

  int aux = difficulty;
  int blocks = (difficulty + 1) / 2;

  *result = 1;
  for(int i = 0; i < blocks; i++) {
    unsigned char cmp;

    if(aux == 1) {
      cmp = 0x0F;
    } else {
      cmp = 0x00;
    }

    if(out[i] > cmp && out[SHA256_BLOCK_SIZE - 1 - i] > cmp)
      *result = -1;

    aux -= 2;
  }
}

void pre_sha256() {
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

void cuda_sha256(string str, int difficulty) {
  BYTE *buff = reinterpret_cast<unsigned char*>(const_cast<char*>(str.c_str()));

  int size = str.length();
  int res;
  BYTE *in, *out;
  int *result;

  hipMalloc((void **)&in, size);
  hipMalloc((void **)&out, SHA256_BLOCK_SIZE);
  hipMalloc((void **)&result, sizeof(int));

  hipMemcpy(in, buff, size * sizeof(BYTE), hipMemcpyHostToDevice);

  pre_sha256();
  sha256_cuda <<< 1, 1 >>> (in, out, size, difficulty, result);

  hipDeviceSynchronize();

  hipMemcpy(&res, result, sizeof(int), hipMemcpyDeviceToHost);

  cout << "result = " << res << endl;

  hipFree(in);
  hipFree(out);
  hipFree(result);
}