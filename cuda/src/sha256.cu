#include "hip/hip_runtime.h"
#include<sha256.cuh>

__global__ void sha256_cuda(BYTE *in, BYTE *out, int size) {
  SHA256_CTX ctx;
  sha256_init(&ctx);
  sha256_update(&ctx, in, size);
  sha256_final(&ctx, out);
  printf("\n");
  for (int i = 0; i < SHA256_BLOCK_SIZE; i++) {
    printf("%.2x", out[i]);
  }
  printf("\n");
}

void pre_sha256() {
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

void cuda_sha256() {
  std::string str = "lol";
  BYTE *buff = reinterpret_cast<unsigned char*>(const_cast<char*>(str.c_str()));
  BYTE *result = (BYTE *) malloc(SHA256_BLOCK_SIZE);

  int size = str.length();
  BYTE *in, *out;

  hipMalloc((void **)&in, size);
  hipMalloc((void **)&out, SHA256_BLOCK_SIZE);

  hipMemcpy(in, buff, size * sizeof(BYTE), hipMemcpyHostToDevice);

  pre_sha256();
  sha256_cuda <<< 1, 1 >>> (in, out, size);

  hipDeviceSynchronize();
  hipMemcpy(result, out, SHA256_BLOCK_SIZE, hipMemcpyDeviceToHost);
  printf("result = %s\n", result);

  char * string = (char *)malloc(70);
  int k, i;
  for (i = 0, k = 0; i < SHA256_BLOCK_SIZE; i++, k+= 2) {
    sprintf(string + k, "%.2x", result[i]);
  }
  string[64] = 0;
  
  printf("hash = %s\n", string);
  free(string);
  hipFree(in);
  hipFree(out);
  
}